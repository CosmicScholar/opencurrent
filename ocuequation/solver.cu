/*
 *  Copyright 2008-2009 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "hip/hip_runtime.h"

#include <cstdio>
#include "ocuutil/timing_pool.h"
#include "ocuequation/solver.h"

namespace ocu {





Solver::~Solver()
{
}


Solver::Solver() {

}

bool Solver::PostKernelDim(const char *kernel_name, dim3 grid, dim3 block)
{
  if (!_wrapper.PostKernelDim(kernel_name, grid, block)) {
    add_error();
    return false;
  }
  return true;
}

bool Solver::PostKernelDim(const char *kernel_name, dim3 grid, dim3 block, int resolution)
{
  if (!_wrapper.PostKernelDim(kernel_name, grid, block, resolution)) {
    add_error();
    return false;
  }
  return true;
}


bool Solver::PostKernel(const char *kernel_name) {
  if (!_wrapper.PostKernel(kernel_name)) {
    add_error();
    return false;
  }
  return true;
}

bool Solver::PostKernel(const char *kernel_name, int resolution) {
  if (!_wrapper.PostKernel(kernel_name, resolution)) {
    add_error();
    return false;
  }
  return true;
}

} // end namespace
